
#include <hip/hip_runtime.h>
/* Test: result = thread ID.
 *
 * CUDA equivalent of test_hello_ptx.ml kernel.
 */

__global__ void test(const float* input, float* result, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // if (i < N)
        result[i] = float(i);
}
