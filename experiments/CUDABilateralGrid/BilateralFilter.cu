#include "hip/hip_runtime.h"
#include "BilateralFilter.h"
#include <CUDAVector3D.h>
#include <MathUtils.h>

#include <cutil_math.h>

// TODO: be very careful with this variable
texture< float2, hipTextureType2D, hipReadModeElementType > g_gridTexture( 0, hipFilterModeLinear );

// TODO: pass in reciprocal samplingSpatial, samplingRange, inputRange, etc
// TODO: padding is a constant 3

// TODO: refactor kernels into another file to decouple from C++ interface
__global__
void createGridKernel( DeviceVector2D< float > inputImage,
					  float inputMin, float inputRange,
					  float samplingSpatial, float samplingRange,
					  int paddingXY, int paddingZ,
					  DeviceArray3D< float2 > outputGrid )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= inputImage.width ||
		y >= inputImage.height )
	{
		return;
	}
	
	// load data
	// and normalize to [0,1]
	float lum = inputImage( x, y );
	lum = ( lum - inputMin ) / inputRange;

	int gx = roundToInt( x / samplingSpatial ) + paddingXY;
	int gy = roundToInt( y / samplingSpatial ) + paddingXY;
	int gz = roundToInt( lum / samplingRange ) + paddingZ;

	float2* pOutputPixel = &( outputGrid( gx, gy, gz ) );
	atomicAdd( &( pOutputPixel->x ), lum );
	atomicAdd( &( pOutputPixel->y ), 1 );
}

__global__
void createCrossGridKernel( DeviceVector2D< float > dataImage,
						   DeviceVector2D< float > edgeImage,
						   float edgeMin, float edgeRange,
						   float samplingSpatial, float samplingRange,
						   int paddingXY, int paddingZ,
						   DeviceArray3D< float2 > outputGrid )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= dataImage.width ||
		y >= dataImage.height )
	{
		return;
	}
	
	// load edge term
	// and normalize to [0,1]
	float e = edgeImage( x, y );	
	e = ( e - edgeMin ) / edgeRange;

	int gx = roundToInt( x / samplingSpatial ) + paddingXY;
	int gy = roundToInt( y / samplingSpatial ) + paddingXY;
	int gz = roundToInt( e / samplingRange ) + paddingZ;

	// put pixel from "data" into grid
	// at location dictated by "edge"
	float d = dataImage( x, y );

	float2* pOutputPixel = &( outputGrid( gx, gy, gz ) );
	atomicAdd( &( pOutputPixel->x ), d );
	atomicAdd( &( pOutputPixel->y ), 1 );
}

__global__
void blurGridKernel( DeviceArray3D< float2 > inputGrid,
					int3 delta, // blur direction
					DeviceArray3D< float2 > outputGrid )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int z = blockIdx.z * blockDim.z + threadIdx.z;

	if( x >= inputGrid.width ||
		y >= inputGrid.height ||
		z >= inputGrid.depth )
	{
		return;
	}

	// compute delta
	int3 xyz = make_int3( x, y, z );

	int3 lo = make_int3( 0 );
	int3 hi = make_int3( inputGrid.width - 1, inputGrid.height - 1, inputGrid.depth - 1 );
	
	float2 v0 = inputGrid( clamp( xyz - 2 * delta, lo, hi ) );
	float2 v1 = inputGrid( clamp( xyz - delta, lo, hi ) );
	float2 v2 = inputGrid( clamp( xyz, lo, hi ) );
	float2 v3 = inputGrid( clamp( xyz + delta, lo, hi ) );
	float2 v4 = inputGrid( clamp( xyz + 2 * delta, lo, hi ) );

	float2 sum =
	(
		0.135335283236613f * v0 +
		0.606530659712633f * v1 +
		v2 + 
		0.606530659712633f * v3 +
		0.135335283236613f * v4
	);

	// early division on the last pass
	// TODO: use a separate pass? 
	if( delta.z == 1 )
	{
		if( sum.y == 0 )
		{
			outputGrid( x, y, z ) = make_float2( 0, 0 );
		}
		else
		{
			outputGrid( x, y, z ) = make_float2( sum.x / sum.y, 0 );
		}
	}
	else
	{
		outputGrid( x, y, z ) = sum;
	}
}

__global__
void sliceGridKernel( DeviceVector2D< float > inputImage,
					 int gridWidth, int gridHeight,
					 float inputMin, float inputRange,
					 float samplingSpatial, float samplingRange,
					 int paddingXY, int paddingZ,
					 DeviceVector2D< float > outputImage )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= inputImage.width ||
		y >= inputImage.height )
	{
		return;
	}
	
	// load data
	// and normalize to [0,1]
	float lum = inputImage( x, y );
	lum = ( lum - inputMin ) / inputRange;

	float gx = x / samplingSpatial + paddingXY;
	float gy = y / samplingSpatial + paddingXY;
	float gz = lum / samplingRange + paddingZ;

	int gz0 = static_cast< int >( gz );
	int gz1 = gz0 + 1;
	float gzFrac = gz - gz0;	

	float v0 = tex2D( g_gridTexture, gx, gz0 * gridHeight + gy ).x;
	float v1 = tex2D( g_gridTexture, gx, gz1 * gridHeight + gy ).x;
	
	float v = lerp( v0, v1, gzFrac );
	outputImage( x, y ) = v;
}

BilateralFilter::BilateralFilter( int imageWidth, int imageHeight,
								 float sigmaSpatial, float sigmaRange,
								 float edgeMin, float edgeMax,
								 bool cross )
{
	setup( imageWidth, imageHeight,
		sigmaSpatial, sigmaRange,
		edgeMin, edgeMax,
		cross );
}

void BilateralFilter::setup( int imageWidth, int imageHeight,
							float sigmaSpatial, float sigmaRange,
							float edgeMin, float edgeMax,
							bool cross )
{
	md_dataImage.resize( imageWidth, imageHeight );
	if( cross )
	{
		md_edgeImage.resize( imageWidth, imageHeight );
	}

	md_outputImage.resize( imageWidth, imageHeight );	

	/*
	// If we allowed variable sampling rates,
	// we'd need to compute derived sigma parameters
	// (gaussian sigmas in the downsampled space)
	float derivedSigmaSpatial = sigmaSpatial / samplingSpatial;
	float derivedSigmaRange = sigmaRange / samplingRange;

	m_paddingXY = static_cast< int >( 2 * derivedSigmaSpatial ) + 1;
	m_paddingZ = static_cast< int >( 2 * derivedSigmaRange ) + 1;
	*/

	m_samplingSpatial = sigmaSpatial;
	m_samplingRange = sigmaRange;

	m_edgeMin = edgeMin;
	m_edgeRange = edgeMax - edgeMin;

	m_paddingXY = 3;
	m_paddingZ = 3;

	// allocate 3D grid	
	int gridWidth = static_cast< int >( ( imageWidth - 1 ) / m_samplingSpatial ) + 1 + 2 * m_paddingXY;
	int gridHeight = static_cast< int >( ( imageHeight - 1 ) / m_samplingSpatial ) + 1 + 2 * m_paddingXY;
	int gridDepth = static_cast< int >( m_edgeRange / m_samplingRange ) + 1 + 2 * m_paddingZ;

	md_grids[0].resize( gridWidth, gridHeight, gridDepth );
	md_grids[1].resize( gridWidth, gridHeight, gridDepth );
}

void BilateralFilter::setInput(const Array2D< float >& input)
{
	md_dataImage.copyFromHost( input );
}

void BilateralFilter::getOutput(Array2D< float >& output)
{
	md_outputImage.copyToHost( output );
}

void BilateralFilter::apply(  )
{
	// input --> md_dataImage
	//md_dataImage.copyFromHost( input );

	createGrid();

	blurGrid();
	
	sliceGrid();

	// md_outputImage --> output
	//md_outputImage.copyToHost( output );
}

void BilateralFilter::applyCross( const Array2D< float >& data, const Array2D< float >& edge,
	Array2D< float >& output )
{
	md_dataImage.copyFromHost( data );
	md_edgeImage.copyFromHost( edge );

	createCrossGrid();

	blurGrid();
	
	sliceCrossGrid();

	// md_outputImage --> output
	md_outputImage.copyToHost( output );
}

void BilateralFilter::createGrid()
{
	// TODO: launching a kernel to do a clear
	// might actually be faster than a hipMemset( 0 )
	md_grids[0].clear();

	dim3 blockDim( 16, 16, 1 );

	int gx = numBins( md_dataImage.width(), blockDim.x );
	int gy = numBins( md_dataImage.height(), blockDim.y );
	dim3 gridDim( gx, gy, 1 );

	createGridKernel<<< gridDim, blockDim >>>
	(
		md_dataImage.deviceVector(),
		m_edgeMin, m_edgeRange,
		m_samplingSpatial, m_samplingRange,
		m_paddingXY, m_paddingZ,
		md_grids[0].deviceArray()
	);
}

void BilateralFilter::createCrossGrid()
{
	// TODO: launching a kernel to do a clear
	// might actually be faster than a hipMemset( 0 )
	md_grids[0].clear();

	dim3 blockDim( 16, 16, 1 );

	int gx = numBins( md_dataImage.width(), blockDim.x );
	int gy = numBins( md_dataImage.height(), blockDim.y );
	dim3 gridDim( gx, gy, 1 );

	createCrossGridKernel<<< gridDim, blockDim >>>
	(
		md_dataImage.deviceVector(),
		md_edgeImage.deviceVector(),
		m_edgeMin, m_edgeRange,
		m_samplingSpatial, m_samplingRange,
		m_paddingXY, m_paddingZ,
		md_grids[0].deviceArray()
	);
}

void BilateralFilter::blurGrid()
{
	Array3D< float2 > tmp( md_grids[0].width(), md_grids[0].height(), md_grids[0].depth() );

	dim3 blockDim( 8, 8, 8 );
    //dim3 blockDim( 8, 8, 1 ); // jrk

	int gx = numBins( md_grids[0].width(), blockDim.x );
	int gy = numBins( md_grids[0].height(), blockDim.y );
	int gz = numBins( md_grids[0].depth(), blockDim.z );
	dim3 gridDim( gx, gy, gz );

	// blur x
	blurGridKernel<<< gridDim, blockDim >>>
	(
		md_grids[0].deviceArray(),
		make_int3( 1, 0, 0 ),
		md_grids[1].deviceArray()
	);

	// blur y
	blurGridKernel<<< gridDim, blockDim >>>
	(
		md_grids[1].deviceArray(),
		make_int3( 0, 1, 0 ),
		md_grids[0].deviceArray()
	);

	// blur z
	blurGridKernel<<< gridDim, blockDim >>>
	(
		md_grids[0].deviceArray(),
		make_int3( 0, 0, 1 ),
		md_grids[1].deviceArray()
	);
}

void BilateralFilter::sliceGrid()
{
	size_t offset;
	hipChannelFormatDesc cd = hipCreateChannelDesc< float2 >();
	void* ptr = md_grids[1].pitchedPointer().ptr;

	hipBindTexture2D
	(
		&offset,
		&g_gridTexture,
		ptr,
		&cd,
		md_grids[1].width(),
		md_grids[1].height() * md_grids[1].depth(),
		md_grids[1].rowPitch()
	);

	dim3 blockDim( 16, 16, 1 );
    //dim3 blockDim( m_samplingSpatial, m_samplingSpatial, 1 ); //jrk
    //dim3 blockDim( 8, 8, 1 ); //jrk
	int gx = numBins( md_dataImage.width(), blockDim.x );
	int gy = numBins( md_dataImage.height(), blockDim.y );
	dim3 gridDim( gx, gy, 1 );

	sliceGridKernel<<< gridDim, blockDim >>>
	(
		md_dataImage.deviceVector(),
		md_grids[1].width(), md_grids[1].height(),
		m_edgeMin, m_edgeRange,
		m_samplingSpatial, m_samplingRange,
		m_paddingXY, m_paddingZ,
		md_outputImage.deviceVector()
	);

	hipUnbindTexture( &g_gridTexture );
}

void BilateralFilter::sliceCrossGrid()
{
	size_t offset;
	hipChannelFormatDesc cd = hipCreateChannelDesc< float2 >();
	void* ptr = md_grids[1].pitchedPointer().ptr;

	hipBindTexture2D
	(
		&offset,
		&g_gridTexture,
		ptr,
		&cd,
		md_grids[1].width(),
		md_grids[1].height() * md_grids[1].depth(),
		md_grids[1].rowPitch()
	);

	dim3 blockDim( 16, 16, 1 );
	int gx = numBins( md_dataImage.width(), blockDim.x );
	int gy = numBins( md_dataImage.height(), blockDim.y );
	dim3 gridDim( gx, gy, 1 );

	sliceGridKernel<<< gridDim, blockDim >>>
	(
		md_edgeImage.deviceVector(),
		md_grids[1].width(), md_grids[1].height(),
		m_edgeMin, m_edgeRange,
		m_samplingSpatial, m_samplingRange,
		m_paddingXY, m_paddingZ,
		md_outputImage.deviceVector()
	);

	hipUnbindTexture( &g_gridTexture );
}
